#include "hip/hip_runtime.h"
﻿#include "fractal.h"

template <typename D, typename R> [[nodiscard]] constexpr auto&& debug_release_gpu([[maybe_unused]] D&& d, [[maybe_unused]] R&& r) noexcept {
#if defined _DEBUG
	return std::forward <D>(d);
#else
	return std::forward <R>(r);
#endif
}


__device__ const std::size_t g_colors_gpu{ debug_release_gpu(64, 256) };
__device__ const real_t      g_infinity_gpu{ 4 };

__device__ pfc::bmp::pixel_t iterate_gpu(complex_t_gpu const c) {
	std::size_t i{};
	complex_t_gpu   z{};

	while ((i++ < g_colors_gpu) && (pfc::norm(z) < g_infinity_gpu))
		z = z * z + c;

	++i;   // gives a nice effect

	pfc::bmp::pixel_t p{};
	return { p.green = static_cast<pfc::byte_t>(1.0 * i / g_colors_gpu * 255) };
}

__global__ void calculate_fractal(pfc::bmp::pixel_t* bmp_data, size_t width, size_t height, complex_t_gpu const ll, double dx, double dy) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height) return;

	float real = ll.real + x * dx;
	float imag = ll.imag + y * dy;
	complex_t_gpu c(real, imag);

	pfc::bmp::pixel_t value = iterate_gpu(c);

	bmp_data[y * width + x] = value;
}

hipError_t calculate_fractal_caller(pfc::bmp::pixel_t* bmp_data, size_t width, size_t height, complex_t_gpu const ll, double dx, double dy) {
	dim3 dimBlock(32, 32);
	dim3 dimGrid(((width + dimBlock.x - 1) / dimBlock.x),
		((height + dimBlock.y - 1) / dimBlock.y));
	
	calculate_fractal <<<dimGrid, dimBlock>>> (bmp_data, width, height, ll, dx, dy);

	return hipGetLastError();
}