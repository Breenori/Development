#include "hip/hip_runtime.h"
﻿#include "./HelloWorld.h"


__global__ void copy_string_kernel(int const size, char * const p_dst, char const * const p_src) {	//kernel function immer void!
	
	auto const t{blockIdx.x*blockDim.x + threadIdx.x}; //thread Number //threadIdx= wievielter Thread vom Block, blockIdx = welcher Block, BlockDim-> Threads pro Block -> x weil Blöcke zwischen 1 und 3 Dimensionen haben können
					
	//für bitmap eig sehr praktisch, ist 2 Dimensional und wenn man Blöcke 2 Dimensional anordnert kann man wieder wie hier arbeiten -> direkt zuweisen
	if(t < size)
		p_dst[t] = p_src[t];


}



void copy_string_kernel_caller(char* const p_dst, char const* const p_src, integer const size) {		//kernel aufrufen
	integer const tib{ 32 }; // threads in block
	integer const big{ (size + tib - 1) / tib }; // blocks in grid
	
	copy_string_kernel <<< big, tib >>> (size, p_dst, p_src); //<<<1, size>>> kernel configuration -> 1 Block mit size Menge an Threads
}








